// Name: Device_Props.cu
// Author: Cody R. Drisko (crdrisko)
// Date: 06/22/2019-15:45:27
// Description: USing device properties


#include <stdio.h>
#include <hip/driver_types.h>
#include "hip/hip_runtime.h"
#include "../../common/book.h"

int main ( void )
{
	hipDeviceProp_t prop;
	int dev;

	HANDLE_ERROR ( hipGetDevice ( &dev ) );
	printf ( "ID of current CUDA device: %d\n", dev );

	memset ( &prop, 0, sizeof ( hipDeviceProp_t ) );
	prop.major = 1;
	prop.minor = 3;
	HANDLE_ERROR ( hipChooseDevice ( &dev, &prop ) );
	printf ( "ID of CUDA device closest to revision 1.3: %d\n", dev );
	HANDLE_ERROR ( hipSetDevice ( dev ) );
}