// Name: Kernel_Call.cu
// Author: Cody R. Drisko (crdrisko)
// Date: 06/22/2019-15:30:42
// Description: A complete "Hello, World!" script run on the device

#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void kernel ( void ) {}

int main ( void )
{
	kernel <<< 1, 1 >>> ( );
	printf ( "Hello, World!\n" );
	return 0;
}
